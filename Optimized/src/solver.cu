#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/solver.h"
#include "../include/cudakernels.h"

using namespace std;

Solver::Solver( vector<double*> d_value, vector<size_t*> d_index, vector<size_t> max_row_size, 
                vector<double*> d_p_value, vector<size_t*> d_p_index, vector<size_t> p_max_row_size, 
                vector<double*> d_r_value, vector<size_t*> d_r_index, vector<size_t> r_max_row_size, 
                size_t numLevels, vector<size_t> num_rows, double damp, 
                size_t* &d_num_rows, size_t* &d_max_row_size, size_t* &d_p_max_row_size, size_t* &d_r_max_row_size)
: m_d_value(d_value), m_d_index(d_index), m_max_row_size(max_row_size), m_d_p_value(d_p_value), m_d_p_index(d_p_index), m_p_max_row_size(p_max_row_size), m_d_r_value(d_r_value), m_d_r_index(d_r_index), m_r_max_row_size(r_max_row_size), m_numLevels(numLevels), m_num_rows(num_rows), m_damp(damp) 
{    
    m_d_num_rows = d_num_rows;
    m_d_max_row_size = d_max_row_size;
    m_d_p_max_row_size = d_p_max_row_size;
    m_d_r_max_row_size = d_r_max_row_size;
}

void Solver::set_verbose(bool verbose, bool bs_verbose) { m_verbose = verbose; m_bs_verbose = bs_verbose; }

// DEBUG:
void Solver::set_steps(size_t step, size_t bs_step)
{
    m_step = step;
    m_bs_step = bs_step;
}


void Solver::set_num_prepostsmooth(size_t pre_n, size_t post_n)
{
    m_numPreSmooth = pre_n;
    m_numPostSmooth = post_n;
}

void Solver::set_convergence_params( size_t maxIter, double minRes, double minRed )
{
	m_maxIter = maxIter;
	m_minRes = minRes;
	m_minRed = minRed;
}

void Solver::set_convergence_params_( size_t maxIter, size_t bs_maxIter, double minRes, double minRed )
{
	m_maxIter = maxIter;
	m_bs_maxIter = bs_maxIter;
	m_minRes = minRes;
	m_minRed = minRed;
}


void Solver::set_bs_convergence_params( size_t maxIter, double minRes, double minRed )
{
	m_bs_maxIter = maxIter;
	m_bs_minRes = minRes;
	m_bs_minRed = minRed;
}



// TODO: could try as a destructor
Solver::~Solver()
{
    // // cout << "solver : deallocate" << endl;
    // CUDA_CALL( hipFree(m_d_res0) );
    // CUDA_CALL( hipFree(m_d_res) );
    // CUDA_CALL( hipFree(m_d_lastRes) );
    // CUDA_CALL( hipFree(m_d_minRes) );
    // CUDA_CALL( hipFree(m_d_minRed) );
    // CUDA_CALL( hipFree(m_d_r) );
    // CUDA_CALL( hipFree(m_d_c) );
    // CUDA_CALL( hipFree(m_d_step) );
    // CUDA_CALL( hipFree(m_d_bs_step) );
    
    // // base solver
    // CUDA_CALL( hipFree(m_d_bs_r) );
    // CUDA_CALL( hipFree(m_d_bs_z) );
    // CUDA_CALL( hipFree(m_d_bs_res) );
    // CUDA_CALL( hipFree(m_d_bs_lastRes) );
    // CUDA_CALL( hipFree(m_d_bs_res0) );
    // // CUDA_CALL( hipFree(m_d_bs_minRes) );
    // // CUDA_CALL( hipFree(m_d_bs_minRed) );
    // CUDA_CALL( hipFree(m_d_bs_rho_old) );
    // CUDA_CALL( hipFree(m_d_bs_p) );
    // CUDA_CALL( hipFree(m_d_bs_alpha) );
    // CUDA_CALL( hipFree(m_d_bs_alpha_temp) );
    
}




void Solver::set_cycle(const char type)
{
    switch(type){
        case 'V': m_gamma = 1; break;
        case 'W': m_gamma = 2; break;
        case 'F': m_gamma = -1; break;
        
        default: std::cout << "Cycle type '" << type << "' invalid argument" << std::endl;
        throw std::invalid_argument("Cycle type: invalid argument");
    }
}
    
bool Solver::init()
{

        m_topLev = m_numLevels - 1;

        // convergence checks
        CUDA_CALL( hipMalloc((void**)&m_d_foo, sizeof(bool)) );
        CUDA_CALL( hipMemcpy(m_d_foo, &m_foo, sizeof(bool), hipMemcpyHostToDevice) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_foo, sizeof(bool)) );
        CUDA_CALL( hipMemcpy(m_d_bs_foo, &m_bs_foo, sizeof(bool), hipMemcpyHostToDevice) );

        CUDA_CALL( hipMalloc((void**)&aps, sizeof(double) * m_p_max_row_size[0]) );
        
        
        // CUDA_CALL( hipMalloc((void**)&m_d_num_rows, sizeof(size_t) * m_numLevels) );
        // for ( int i = 0; i < m_numLevels ; i++)
        //     CUDA_CALL( hipMemcpy(&m_d_num_rows[i], &m_num_rows[i], sizeof(size_t), hipMemcpyHostToDevice) );
        
        // CUDA_CALL( hipMalloc((void**)&m_d_max_row_size, sizeof(size_t) * m_numLevels) );
        // for ( int i = 0; i < m_numLevels ; i++)
        //     CUDA_CALL( hipMemcpy(&m_d_max_row_size[i], &m_max_row_size[i], sizeof(size_t), hipMemcpyHostToDevice) );
            
        // CUDA_CALL( hipMalloc((void**)&m_d_p_max_row_size, sizeof(size_t) * (m_numLevels - 1)) );
        // for ( int i = 0; i < m_numLevels - 1 ; i++)
        //     CUDA_CALL( hipMemcpy(&m_d_p_max_row_size[i], &m_p_max_row_size[i], sizeof(size_t), hipMemcpyHostToDevice) );
        
        // CUDA_CALL( hipMalloc((void**)&m_d_r_max_row_size, sizeof(size_t) * (m_numLevels - 1)) );
        // for ( int i = 0; i < m_numLevels - 1; i++)
        //     CUDA_CALL( hipMemcpy(&m_d_r_max_row_size[i], &m_r_max_row_size[i], sizeof(size_t), hipMemcpyHostToDevice) );
        


        // calculate cuda grid and block dimensions of each level
        m_gridDim.resize(m_numLevels);
        m_blockDim.resize(m_numLevels);

        for ( int i = 0 ; i < m_numLevels ; i++ )
            calculateDimensions(m_num_rows[i], m_gridDim[i], m_blockDim[i]);
		    
		m_gridDim_cols.resize(m_numLevels - 1);
        m_blockDim_cols.resize(m_numLevels - 1);

        for ( int i = 0 ; i < m_numLevels - 1; i++ )
		    calculateDimensions(m_num_rows[i], m_gridDim_cols[i], m_blockDim_cols[i]);

        
        CUDA_CALL( hipMalloc((void**)&m_d_r, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMemset(m_d_r, 0, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMalloc((void**)&m_d_c, sizeof(double) * m_num_rows[m_topLev]) );
        CUDA_CALL( hipMemset(m_d_c, 0, sizeof(double) * m_num_rows[m_topLev]) );

        
        // temp residuum
        CUDA_CALL( hipMalloc((void**)&m_d_res0, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_res0, 0, sizeof(double)) );

        // last residuum
        CUDA_CALL( hipMalloc((void**)&m_d_lastRes, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_lastRes, 0, sizeof(double)) );
        
        // current residuum
        CUDA_CALL( hipMalloc((void**)&m_d_res, sizeof(double)) );
        CUDA_CALL( hipMemset(m_d_res, 0, sizeof(double)) );
    
        // minimum required residuum for convergence
        // d_m_minRes;
        CUDA_CALL( hipMalloc((void**)&m_d_minRes, sizeof(double)) );
        CUDA_CALL( hipMemcpy(m_d_minRes, &m_minRes, sizeof(double), hipMemcpyHostToDevice) );
        
        // minimum required reduction for convergence
        // d_m_minRed;
        CUDA_CALL( hipMalloc((void**)&m_d_minRed, sizeof(double)) );
        CUDA_CALL( hipMemcpy(m_d_minRed, &m_minRed, sizeof(double), hipMemcpyHostToDevice) );
        
        // steps
        CUDA_CALL( hipMalloc((void**)&m_d_step, sizeof(size_t)) );
        CUDA_CALL( hipMemset(m_d_step, 0, sizeof(size_t)) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_step, sizeof(size_t)) );
        CUDA_CALL( hipMemset(m_d_bs_step, 0, sizeof(size_t)) );
        
        /// GMG precond
        // residuum and correction vectors on each level
        m_d_gmg_r.resize(m_numLevels);
        m_d_gmg_c.resize(m_numLevels);
        
        // temporary residuum vectors for GMG
        m_d_rtmp.resize(m_numLevels);
        
        // temporary correction vectors for GMG
        m_d_ctmp.resize(m_numLevels);
        
        for ( int i = 0 ; i < m_numLevels ; i++ )
        {
            CUDA_CALL( hipMalloc((void**)&m_d_gmg_r[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMalloc((void**)&m_d_gmg_c[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_gmg_r[i], 0, sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_gmg_c[i], 0, sizeof(double) * m_num_rows[i] ) );
            
            CUDA_CALL( hipMalloc((void**)&m_d_rtmp[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_rtmp[i], 0, sizeof(double) * m_num_rows[i] ) );
            
            CUDA_CALL( hipMalloc((void**)&m_d_ctmp[i], sizeof(double) * m_num_rows[i] ) );
            CUDA_CALL( hipMemset(m_d_ctmp[i], 0, sizeof(double) * m_num_rows[i] ) );
        }
        
        
        // base-solver
        
        CUDA_CALL( hipMalloc((void**)&m_d_bs_r, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_r, 0, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_z, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_z, 0, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_p, sizeof(double) * m_num_rows[0] ) );
        CUDA_CALL( hipMemset(m_d_bs_p, 0, sizeof(double) * m_num_rows[0] ) );
        
        CUDA_CALL( hipMalloc((void**)&m_d_bs_res, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_res, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_res0, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_res0, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_lastRes, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_lastRes, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_rho, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_rho, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_rho_old, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_rho_old, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_alpha, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_alpha, 0, sizeof(double) ) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_alpha_temp, sizeof(double) ) );
        CUDA_CALL( hipMemset(m_d_bs_alpha_temp, 0, sizeof(double) ) );

        CUDA_CALL( hipMalloc((void**)&m_d_maxIter, sizeof(size_t)) );
        CUDA_CALL( hipMemcpy(m_d_maxIter, &m_maxIter, sizeof(size_t), hipMemcpyHostToDevice) );
        CUDA_CALL( hipMalloc((void**)&m_d_bs_maxIter, sizeof(size_t)) );
        CUDA_CALL( hipMemcpy(m_d_bs_maxIter, &m_bs_maxIter, sizeof(size_t), hipMemcpyHostToDevice) );
        
       

    return true;
}

bool Solver::reinit()
{
        
        setToZero_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( m_d_r, &m_d_num_rows[m_topLev] );
        setToZero_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( m_d_c, &m_d_num_rows[m_topLev] );

        for ( int lev = 0 ; lev < m_numLevels ; lev++ )
        {
            setToZero_<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_gmg_r[lev], &m_d_num_rows[lev] );
            setToZero_<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_gmg_c[lev], &m_d_num_rows[lev] );
            setToZero_<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_rtmp[lev], &m_d_num_rows[lev] );
            setToZero_<<<m_gridDim[lev], m_blockDim[lev]>>>( m_d_ctmp[lev], &m_d_num_rows[lev] );
        }

        // scalars
        setToZero<<<1, 1>>>( m_d_res0, 1 );
        setToZero<<<1, 1>>>( m_d_lastRes, 1 );
        setToZero<<<1, 1>>>( m_d_res, 1 );
        setToZero<<<1, 1>>>( m_d_step, 1 );
        setToZero<<<1, 1>>>( m_d_bs_step, 1 );


        // base-solver
        setToZero_<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_r, &m_d_num_rows[0] );
        setToZero_<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_z, &m_d_num_rows[0] );
        setToZero_<<<m_gridDim[0], m_blockDim[0]>>>( m_d_bs_p, &m_d_num_rows[0] );
        setToZero<<<1, 1>>>( m_d_bs_res, 1 );
        setToZero<<<1, 1>>>( m_d_bs_res0, 1 );
        setToZero<<<1, 1>>>( m_d_bs_lastRes, 1 );
        setToZero<<<1, 1>>>( m_d_bs_rho, 1 );
        setToZero<<<1, 1>>>( m_d_bs_rho_old, 1 );
        setToZero<<<1, 1>>>( m_d_bs_alpha, 1 );
        setToZero<<<1, 1>>>( m_d_bs_alpha_temp, 1 );

        

        return true;
}

bool Solver::precond(double* m_d_c, double* m_d_r)
{
    
    // reset correction
    // c.resize(d.size()); 
    // c = 0.0;
	setToZero<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_c, m_num_rows[m_topLev]);

    // Vector<double> rtmp(r);
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_rtmp[m_topLev], m_d_r, m_num_rows[m_topLev]);
    
	// NOTE: the original d_c and d_r from i_s.cu stay here
	// d_gmg_c[topLev] = d_c
	// d_gmg_r[topLev] = d_r
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_gmg_c[m_topLev], m_d_c, m_num_rows[m_topLev]);
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_gmg_r[m_topLev], m_d_r, m_num_rows[m_topLev]);

    precond_add_update_GPU(m_d_gmg_c[m_topLev], m_d_rtmp[m_topLev], m_topLev, m_gamma);

    vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_c, m_d_gmg_c[m_topLev], m_num_rows[m_topLev]);
	vectorEquals_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_r, m_d_gmg_r[m_topLev], m_num_rows[m_topLev]);

    return true;
}

// A*c = r ==> A_coarse*d_bs_u = d_bs_b
bool Solver::base_solve(double* d_bs_u, double* d_bs_b)
{

    // resetting base solver variables to zero
    setToZero_<<<1,m_num_rows[0]>>>(m_d_bs_r, &m_d_num_rows[0]);
    setToZero_<<<1,m_num_rows[0]>>>(m_d_bs_p, &m_d_num_rows[0]);
    setToZero_<<<1,m_num_rows[0]>>>(m_d_bs_z, &m_d_num_rows[0]);
    setToZero<<<1,1>>>(m_d_bs_rho, 1);
    setToZero<<<1,1>>>(m_d_bs_rho_old, 1);
    setToZero<<<1,1>>>(m_d_bs_alpha, 1);
    setToZero<<<1,1>>>(m_d_bs_alpha_temp, 1);
    setToZero<<<1,1>>>(m_d_bs_res, 1);
    setToZero<<<1,1>>>(m_d_bs_res0, 1);
    setToZero<<<1,1>>>(m_d_bs_lastRes, 1);
    setToZero<<<1,1>>>(m_d_bs_step, 1);
    setToTrue<<<1,1>>>(m_d_bs_foo);
    m_bs_foo = true;

       

    // m_d_bs_r = d_bs_b - A*d_bs_u
    ComputeResiduum_GPU_<<<m_gridDim[0],m_blockDim[0]>>>(&m_d_num_rows[0], &m_d_max_row_size[0], m_d_value[0], m_d_index[0], d_bs_u, m_d_bs_r, d_bs_b);

    // norm_GPU(m_d_bs_res, m_d_bs_r, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
    norm_GPU_<<<m_gridDim[0], m_blockDim[0]>>>(m_d_bs_res, m_d_bs_r, &m_d_num_rows[0]);

    equals_GPU<<<1,1>>>(m_d_bs_res0, m_d_bs_res);
    
    if ( m_bs_verbose )
    {
        cout << "CG  : ";
        hipDeviceSynchronize();
        printInitialResult_GPU<<<1,1>>>(m_d_bs_res0, m_d_minRes, m_d_minRed);
        hipDeviceSynchronize();
    }
	
 
    // check iteration conditions before the iteration loop
    checkIterationConditions<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_d_bs_res, m_d_bs_res0, m_d_minRes, m_d_minRed, m_bs_maxIter);
    CUDA_CALL( hipMemcpy( &m_bs_foo, m_d_bs_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
    
    if (!m_bs_foo) return true;

    else
    {
        addStep<<<1,1>>>(m_d_bs_step);

        // iteration loop
        int bs_step = 1;

        while(m_bs_foo == true || bs_step < m_bs_maxIter)
        {

            // precond
            Jacobi_Precond_GPU<<<m_gridDim[0], m_blockDim[0]>>>(m_d_bs_z, m_d_value[0], m_d_index[0], m_max_row_size[0], m_d_bs_r, m_num_rows[0], m_damp);

            // z = r
            // vectorEquals_GPU_<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_z, m_d_bs_r, &m_d_num_rows[0]);


            // rho = < z, r >
            dotProduct(m_d_bs_rho, m_d_bs_r, m_d_bs_z, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
         
            // calculate p
            calculateDirectionVector<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_step, m_d_bs_p, m_d_bs_z, m_d_bs_rho, m_d_bs_rho_old, m_num_rows[0]);
    
            /// z = A*p
            Apply_GPU<<<m_gridDim[0],m_blockDim[0]>>>( m_num_rows[0], m_max_row_size[0], m_d_value[0], m_d_index[0], m_d_bs_p, m_d_bs_z );


            // alpha = rho / (p * z)
            calculateAlpha(m_d_bs_alpha, m_d_bs_rho, m_d_bs_p, m_d_bs_z, m_d_bs_alpha_temp, m_num_rows[0], m_gridDim[0], m_blockDim[0] );

            // add correction to solution
            // u = u + alpha * p
            axpy_GPU<<<m_gridDim[0],m_blockDim[0]>>>(d_bs_u, m_d_bs_alpha, m_d_bs_p, m_num_rows[0]);


            // update residuum
            // r = r - alpha * z
            axpy_neg_GPU<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_r, m_d_bs_alpha, m_d_bs_z, m_num_rows[0]);


            // compute residuum
            // lastRes = res;
            equals_GPU<<<1,1>>>(m_d_bs_lastRes, m_d_bs_res);


            // res = r.norm();
            norm_GPU(m_d_bs_res, m_d_bs_r, m_num_rows[0], m_gridDim[0], m_blockDim[0]);
   
    
            // store old rho
            // rho_old = rho;
            vectorEquals_GPU_<<<m_gridDim[0],m_blockDim[0]>>>(m_d_bs_rho_old, m_d_bs_rho, &m_d_num_rows[0]);


            if ( m_bs_verbose )
            {
                cout << "CG  : ";
                hipDeviceSynchronize();
                printResult_GPU<<<1,1>>>(m_d_bs_step, m_d_bs_res, m_d_minRes, m_d_bs_lastRes, m_d_bs_res0, m_d_minRed);
                hipDeviceSynchronize();
            }

            // checkIterationConditionsBS<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_bs_maxIter, m_d_bs_res, m_d_minRes);
            checkIterationConditionsBS_<<<1,1>>>(m_d_bs_foo, m_d_bs_step, m_d_bs_maxIter, m_d_bs_res, m_d_minRes);
            CUDA_CALL( hipMemcpy( &m_bs_foo, m_d_bs_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
            if (!m_bs_foo) break;


                    

            addStep<<<1,1>>>(m_d_bs_step);
    
            bs_step++;
    
        }

        return true;
    
    }
}


bool Solver::precond_add_update_GPU(double* d_c, double* d_r, std::size_t lev, int cycle)
{

    // initialize ctmp[lev] to zero
    setToZero_<<< m_gridDim[lev], m_blockDim[lev] >>>( m_d_ctmp[lev], &m_d_num_rows[lev] );			

    // if on base level
	if( lev == 0 )
	{
        base_solve(m_d_ctmp[lev], d_r);   

        // c += ctmp;
		addVector_GPU_<<< m_gridDim[lev], m_blockDim[lev] >>>(d_c, m_d_ctmp[lev], &m_d_num_rows[0]);

        // r = r - A[0] * ctmp0
		UpdateResiduum_GPU_<<< m_gridDim[lev], m_blockDim[lev] >>>(&m_d_num_rows[lev], &m_d_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);

        return true;
    }

    // presmooth
    for ( int i = 0 ; i < m_numPreSmooth ; i++)
    {
        smoother( m_d_ctmp[lev], d_r, lev );

        // c += ctmp;
        addVector_GPU_<<<m_gridDim[lev], m_blockDim[lev]>>>( d_c, m_d_ctmp[lev], &m_d_num_rows[lev] );
        
        // r -= A[lev] * ctmp;
        UpdateResiduum_GPU_<<< m_gridDim[lev], m_blockDim[lev] >>>(&m_d_num_rows[lev], &m_d_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);

    }

    
    // restrict defect
    setToZero_<<<m_gridDim_cols[lev-1],m_blockDim_cols[lev-1]>>>( m_d_gmg_r[lev-1], &m_d_num_rows[lev-1] );
        
    
    // r_coarse = P^T * r   
    ApplyTransposed_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>(m_num_rows[lev], m_p_max_row_size[lev-1], m_d_p_value[lev-1], m_d_p_index[lev-1], d_r, m_d_gmg_r[lev-1]);

    setToZero_<<<m_gridDim_cols[lev-1],m_blockDim_cols[lev-1]>>>( m_d_gmg_c[lev-1], &m_d_num_rows[lev-1] );

    
    // F-cycle
    if(cycle == -1) 
	{
        // one F-Cycle ...
        if( !precond_add_update_GPU(m_d_ctmp[lev-1], m_d_rtmp[lev-1], lev-1, -1) )  // TODO: check ctmp or gmg_c?
        {
            std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
            return false;
        }

        // ... followed by a V-Cycle
        if( !precond_add_update_GPU(m_d_ctmp[lev-1], m_d_rtmp[lev-1], lev-1, 1) )
        {
            std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
            return false;
        }
	}

    // V- and W-cycle
    else
	{
		for (int g = 0; g < cycle; ++g)
		{
			if( !precond_add_update_GPU(m_d_gmg_c[lev-1], m_d_gmg_r[lev-1], lev-1, cycle) )
			{
				std::cout << "gmg failed on level " << lev << ". Aborting." << std::endl;
				return false;
			}
		
		}
    }
    
    /// prolongate coarse grid correction
	// ctmp = P[lev-1] * c_coarse;
    Apply_GPU<<<m_gridDim[lev],m_blockDim[lev]>>>( m_num_rows[lev], m_p_max_row_size[lev-1], m_d_p_value[lev-1], m_d_p_index[lev-1], m_d_gmg_c[lev-1], m_d_ctmp[lev]);
    
    /// add correction and update defect
	// c += ctmp;
	addVector_GPU_<<<m_gridDim[lev],m_blockDim[lev]>>>(d_c, m_d_ctmp[lev], &m_d_num_rows[lev]);
    
    UpdateResiduum_GPU_<<<m_gridDim[lev],m_blockDim[lev]>>>( &m_d_num_rows[lev], &m_d_max_row_size[lev] , m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);
    
    
    // postsmooth
    for ( int i = 0 ; i < m_numPostSmooth ; i++)
    {
        smoother( m_d_ctmp[lev], d_r, lev );

         // c += ctmp;
        addVector_GPU_<<<m_gridDim[lev], m_blockDim[lev]>>>( d_c, m_d_ctmp[lev], &m_d_num_rows[lev] );

        UpdateResiduum_GPU_<<< m_gridDim[lev], m_blockDim[lev] >>>(&m_d_num_rows[lev], &m_d_max_row_size[lev], m_d_value[lev], m_d_index[lev], m_d_ctmp[lev], d_r);

    }
    

    return true;
}

bool Solver::smoother(double* d_c, double* d_r, int lev)
{
        
    Jacobi_Precond_GPU<<<m_gridDim[lev], m_blockDim[lev]>>>(d_c, m_d_value[lev], m_d_index[lev], m_max_row_size[lev], d_r, m_num_rows[lev], m_damp);

    return true;
}




bool Solver::solve(double* d_u, double* d_b, vector<double*> d_value)
{
    
    // initialization
    setToZero_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( d_u, &m_d_num_rows[m_topLev] );
    setToTrue<<<1,1>>>(m_d_foo);
    m_d_value = d_value;
    m_foo = true;
    

    // r = b - A*u
    ComputeResiduum_GPU_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(&m_d_num_rows[m_topLev], &m_d_max_row_size[m_topLev], m_d_value[m_topLev], m_d_index[m_topLev], d_u, m_d_r, d_b);
    
    
    
    // d_res0 = norm(m_d_r)
    // norm_GPU(m_d_res0, m_d_r, m_num_rows[m_topLev], m_gridDim[m_topLev], m_blockDim[m_topLev]);
    norm_GPU<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>(m_d_res0, m_d_r, m_num_rows[m_topLev]);
    

    // res = res0;
    equals_GPU<<<1,1>>>(m_d_res, m_d_res0);	

    if ( m_verbose )
    {
        cout << "GMG : ";
        hipDeviceSynchronize();
        printInitialResult_GPU<<<1,1>>>(m_d_res0, m_d_minRes, m_d_minRed);
        hipDeviceSynchronize();
    }

    addStep<<<1,1>>>(m_d_step);

    // iteration loop
    while(m_foo)
    {
        // GMG-preconditioner
        precond(m_d_c, m_d_r);
        
        // add correction to solution
        // u += c;
        addVector_GPU_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( d_u, m_d_c, &m_d_num_rows[m_topLev] );
        

        // update residuum r = r - A*c
        UpdateResiduum_GPU_<<<m_gridDim[m_topLev], m_blockDim[m_topLev]>>>( &m_d_num_rows[m_topLev], &m_d_max_row_size[m_topLev], m_d_value[m_topLev], m_d_index[m_topLev], m_d_c, m_d_r );


        // store norm of the last residuum
        // lastRes = res;
        equals_GPU<<<1,1>>>(m_d_lastRes, m_d_res);
        

        // compute new residuum norm
        // res = r.norm();
        norm_GPU(m_d_res, m_d_r, m_num_rows[m_topLev], m_gridDim[m_topLev], m_blockDim[m_topLev]);
        

        if ( m_verbose )
        {
        cout << "GMG : ";
        hipDeviceSynchronize();
        printResult_GPU<<<1,1>>>(m_d_step, m_d_res, m_d_minRes, m_d_lastRes, m_d_res0, m_d_minRed);
        hipDeviceSynchronize();
        }

        // checkIterationConditions<<<1,1>>>(m_d_foo, m_d_step, m_d_res, m_d_res0, m_d_minRes, m_d_minRed, m_maxIter);
        checkIterationConditions_<<<1,1>>>(m_d_foo, m_d_step, m_d_res, m_d_res0, m_d_minRes, m_d_minRed, m_d_maxIter);
        CUDA_CALL( hipMemcpy( &m_foo, m_d_foo, sizeof(bool), hipMemcpyDeviceToHost) 	);
        
        addStep<<<1,1>>>(m_d_step);
    
    }



    return true;
}
